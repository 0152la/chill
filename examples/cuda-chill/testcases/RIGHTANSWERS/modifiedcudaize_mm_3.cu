#include "hip/hip_runtime.h"
// this source is derived from CHILL AST originally from file 'mm.c' as parsed by frontend compiler rose

__global__ void kernel_gpu(float *c[1024], float *a[1024], float *b[1024]) {
  int tx = threadIdx.x;
  int by = blockIdx.y;
  int bx = blockIdx.x;
  int j;
  int i;
  for (i = 0; i <= 1023; i += 1) 
    for (j = 0; j <= 1023; j += 1) 
      {
        c[j][i] = c[j][i] + a[bx][i] * b[j][bx];
      }
}
#define N 1024

void normalMM(float c[1024][1024], float a[1024][1024], float b[1024][1024]) {
  float * devI2Ptr;
  float * devI1Ptr;
  float * devO1Ptr;
  hipMalloc((void **)devO1Ptr, 1048576 * sizeof(float));
  hipMalloc((void **)devI1Ptr, 1048576 * sizeof(float));
  hipMemcpy(devI1Ptr, a, 1048576 * sizeof(float), hipMemcpyHostToDevice);
  hipMalloc((void **)devI2Ptr, 1048576 * sizeof(float));
  hipMemcpy(devI2Ptr, b, 1048576 * sizeof(float), hipMemcpyHostToDevice);
  dim3 dimGrid0 = dim3(1024, 1);
  dim3 dimBlock0 = dim3(1);
  kernel_gpu<<<dimGrid0,dimBlock0>>>((float (*)[1024])float * devO1Ptr, (float (*)[1024])float * devI1Ptr, (float (*)[1024])float * devI2Ptr);
  hipMemcpy(c, devO1Ptr, 1048576 * sizeof(float), hipMemcpyDeviceToHost);
  hipFree(devO1Ptr);
  hipFree(devI1Ptr);
  hipFree(devI2Ptr);
}
