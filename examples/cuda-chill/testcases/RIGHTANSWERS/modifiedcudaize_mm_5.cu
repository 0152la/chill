#include "hip/hip_runtime.h"
// this source is derived from CHILL AST originally from file 'mm.c' as parsed by frontend compiler rose

__global__ void kernel_gpu(float *c[1024], float *a[1024], float *b[1024]) {
  int tx= threadIdx.x;
  int by= blockIdx.y;
  int bx= blockIdx.x;
  
    
      
        c[by][bx] = c[by][bx] + a[tx][bx] * b[by][tx];;;;
}
#define N 1024

void normalMM(float c[1024][1024], float a[1024][1024], float b[1024][1024]) {
  float * devI2Ptr;
  float * devI1Ptr;
  float * devO1Ptr;
  hipMalloc((void **)devO1Ptr, 1048576 * sizeof(float));
  hipMalloc((void **)devI1Ptr, 1048576 * sizeof(float));
  hipMemcpy(devI1Ptr, a, 1048576 * sizeof(float), hipMemcpyHostToDevice);
  hipMalloc((void **)devI2Ptr, 1048576 * sizeof(float));
  hipMemcpy(devI2Ptr, b, 1048576 * sizeof(float), hipMemcpyHostToDevice);
  dim3 dimGrid0= dim3(1024, 1024);
  dim3 dimBlock0= dim3(1024);
  kernel_gpu<<<dimGrid0,dimBlock0>>>((float (*)[1024])float * devO1Ptr, (float (*)[1024])float * devI1Ptr, (float (*)[1024])float * devI2Ptr);
  hipMemcpy(c, devO1Ptr, 1048576 * sizeof(float), hipMemcpyDeviceToHost);
  hipFree(devO1Ptr);
  hipFree(devI1Ptr);
  hipFree(devI2Ptr);
}
